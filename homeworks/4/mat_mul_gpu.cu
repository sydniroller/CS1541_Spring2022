#include <algorithm>
#include <assert.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

float* d_A = NULL;	// Pointer to matrix A in device memory
float* d_B = NULL;	// Pointer to matrix B in device memory
float* d_C = NULL;	// Pointer to result matrix C in device memory

double time_memcpy = 0;
double time_compute = 0;	// Timers

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on CPU: Policy 1
/////////////////////////////////////////////////////////////////////////
void mm_cpu(float* C, const float* A, const float* B, int n) {
	for (int i=0; i<n; i++) {
		for (int j=0; j<n; j++) {
                        C[i*n + j] = 0;
			for (int k=0; k<n; k++) {
				C[i*n + j] += A[i*n + k] * B[k*n + j];
			}
		}
	}
}

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on GPU: Policy 2
/////////////////////////////////////////////////////////////////////////
__global__ void mm_gpu(float* C, float* A, float* B, int n)
{
	// TODO:
	// Implement the naive GPU matrix multiplication shown on the lecture slides.
	// Remove the printf below when you don't need it.  It's just for demonstration.
	/*
	for (int i=0; i<n; i++){
		for (int j=0; j<n; j++){
			for (int k=0; k<n; k++){
				C[i*n + j] += A[i*n + k] * B[k*n +j];
			}
		}
	}
	*/
	float Cvalue = 0;
	int i = blockIdx.y * blockDim.y +  threadIdx.y;
	int j = blockIdx.x * blockDim.x +  threadIdx.x;
	for (int k = 0; k < n; ++k)
		Cvalue += A[i * n + k] * B[k * n + j];
	C[i * n + j] = Cvalue;

	//printf("Grid(%d, %d) Block (%d, %d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

/////////////////////////////////////////////////////////////////////////
// Matrix-matrix Multiplication on GPU using shared memory: Policy 3
/////////////////////////////////////////////////////////////////////////
__global__ void mm_gpu_shared(float* C, float* A, float* B, int n)
{
	int block_size = blockDim.x;
	// Shared memory used to store a tile in arrays A and B respectively.
	// We needed to allocate dynamic shared memory because the block size is unknown ahead of time.
        // This looks strange but if you read the below NVIDIA blog entry, it will make sense:
	// https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
        // This is equivalent to the below static shared memory declarations (but it won't work because of block_size).
	// __shared__ float As[block_size * block_size];
	// __shared__ float Bs[block_size * block_size];
	extern __shared__ float As[];
	float *Bs = &As[block_size * block_size];

	// TODO:
	// Implement GPU matrix multiplication using tiling and the above allocated shared memory.
	
	// For reference, you can look at the following 2 codes:
	// 1. A tiled vector-matrix multiplication implementation is available in /resources/gpu_experiments/mat_vec_gpu.cu of this repository.
	// 2. A tiled matrix multiplication for a special case where the "w" (row size of A and column size of B) is small is shown in:
	// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#shared-memory-in-matrix-multiplication-c-ab
	// Note that a tile in the NVIDIA code is 2-dimensional when As and Bs above declares a 1-dimensional array.
	// (Because you cannot declare dynamic 2-dimensional array in C/C++.)
	// That means you will have to use that 1-d array as a 2-d array using the old "row*width + column" translation.
}

/////////////////////////////////////////////////////////////////////////
// Macro to check that CUDA API calls are successful
/////////////////////////////////////////////////////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/////////////////////////////////////////////////////////////////////////
// Allocate device memory and copy from host to device memory
/////////////////////////////////////////////////////////////////////////
void copy_host_to_device(float* A, float* B, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;
	int nBytes;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	// TODO:
	// 1. Allocate GPU memory for d_A, d_B, d_C using cudaMalloc.
	// 2. Copy input arrays A, B to d_A, d_B using cudaMemcpy.
	// 3. Call cudaMemset on array d_C to initialize all elements to 0.

	// allocate gpu memory for d_A, d_B, d_C using cudaMalloc
	int matrix_size = n * n;
	nBytes = matrix_size * sizeof(float);
	hipMalloc((void **) &d_A, nBytes);
	hipMalloc((void **) &d_B, nBytes);
	hipMalloc((void **) &d_C, nBytes);

	// copy input arrays A, B to d_A, d_B using cudaMemcpy
	hipMemcpy(d_A, A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, nBytes, hipMemcpyHostToDevice);

	// call cudaMemset on array d_C to initialize all elements to 0
	hipMemset(d_C, 0, nBytes);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Copy result from device to host memory and free device memory
/////////////////////////////////////////////////////////////////////////
void copy_device_to_host(float* C, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;
	int nBytes;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	// TODO:
	// 1. Copy result array d_C to C using cudaMemcpy.
	// 2. Free memory allocated for d_A, d_B, d_C using cudaFree.

	// copy result array d_C to C using cudaMemcpy
	int matrix_size = n * n;
	nBytes = matrix_size * sizeof(float);
	hipMemcpy(C, d_C, nBytes, hipMemcpyDeviceToHost);

	// free memory allocated for d_A, d_B, d_C using cudaFree
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	if(argc != 5) {printf("Usage: mat_mul_gpu <array size> <block size> <policy (1|2|3)> <(debug|nodebug)>\n") ; exit(2) ;}
	int N = atoi(argv[1]) ;				// array size
	int block_size = atoi(argv[2]) ;	// size of thread block
	int policy = atoi(argv[3]) ;		// 1, 2 or 3
	bool debug = (!strcmp(argv[4], "debug")) ? true : false;

	//printf("\n Array size = %d , block size = %d and policy = %d\n ", N, block_size, policy);

	// set seed for rand()
	srand(2020);

	// allocate host memory for matrix A and vector x
	int matrix_size = N * N;
	float* h_A = (float*)malloc(sizeof(float) * matrix_size);
	float* h_B = (float*)malloc(sizeof(float) * matrix_size);
	float* h_C = (float*)malloc(sizeof(float) * matrix_size);

	// randomly initialize host memory
	for (int i = 0; i < matrix_size; ++i){
		h_A[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < matrix_size; i++){
		h_B[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < matrix_size; i++){
		h_C[i] = 0.0;
	}

	//call the relevant policy 
	if(policy==1){ //CPU baseline
		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		mm_cpu(h_C, h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;
	}
	else if(policy==2){ //CUDA MV multiplication
		copy_host_to_device(h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		// TODO: Call the kernel
		// Call mm_gpu <<< >>> ( ) with the appropriate grid and thread block layouts.
		dim3 dimBlock(block_size, block_size);
		dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
		mm_gpu <<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N); 

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_C, N);
	}
	else if(policy==3){ //CUDA MV multiplication with shared memory
		copy_host_to_device(h_A, h_B, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

                dim3 dimBlock(block_size, block_size);
                dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
		// Additional 3rd parameter "sizeof(float) * block_size * block_size * 2" is used to specify how much shared memory to allocate for a thread block.
		// It's needed because we want to dynamically allocate shared memory according to the given thread block size.
		mm_gpu_shared <<<dimGrid, dimBlock, sizeof(float) * block_size * block_size * 2>>> (d_C, d_A, d_B, N);

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_C, N);
	}

	if (debug) {
		// print selected results (for checking correctness)
		for (int i=0; i<matrix_size; i+=matrix_size/10) { printf(" %10.6f",h_C[i]); }
		printf("\n");
	}
	if(!debug) {
		printf("Memcpy Time: %f seconds\n", time_memcpy);
		printf("Compute Time: %f seconds\n", time_compute);
	}

	// clean up memory allocated  
	free(h_A);
	free(h_B);
	free(h_C);

	hipDeviceReset();
}
